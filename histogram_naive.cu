#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <stdio.h>
#include "histogram_naive.hpp"

__global__
void histogram_thread_element_kernel(int arrayLength, const float* x, int numBins, float firstEdge, float lastEdge, int* outHist)
{
    // Approaches:
    // 1. One thread per element, writing into a single histogram.  I can figure this out now.
    // 2. One thread per bin, reading the entire array.  I can figure this out now.
    // 3. Something else
    
    // histogram_thread_element_kernel: each thread handles one element and writes into one histogram.
    
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx >= arrayLength)
    {
        return;
    }
    
    float binSize = (lastEdge-firstEdge)/numBins;
    int iBin = (x[idx]-firstEdge)/binSize;
    if (iBin < 0)
    {
        iBin = 0;
    }
    if (iBin >= numBins)
    {
        iBin = numBins - 1;
    }
    
    atomicAdd(outHist + iBin, 1);
}



void histogram_thread_element(int arrayLength, const float* x, int numBins, float firstEdge, float lastEdge, int* outHist, int gridSize, int blockSize)
{
    float* d_x;
    int* d_histogram;
    
    checkCudaErrors(hipMalloc(&d_x, arrayLength*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_histogram, numBins*sizeof(int)));
    
    checkCudaErrors(hipMemcpy(d_x, x, arrayLength*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_histogram, 0, numBins*sizeof(int)));
    
    histogram_thread_element_kernel<<<gridSize, blockSize>>>(arrayLength, d_x, numBins, firstEdge, lastEdge, d_histogram);
    checkCudaErrors(hipMemcpy(outHist, d_histogram, numBins*sizeof(float), hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_histogram));
}

__global__
void histogram_thread_bin_kernel(int arrayLength, const float* x, int numBins, float firstEdge, float lastEdge, int* outHist)
{
    // Approaches:
    // 1. One thread per element, writing into a single histogram.  I can figure this out now.
    // 2. One thread per bin, reading the entire array.  I can figure this out now.
    // 3. Something else
    
    // histogram_thread_bin_kernel: each thread handles one bin
    
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx >= numBins)
    {
        return;
    }
    
    float binSize = (lastEdge-firstEdge)/numBins;
    
    int myBinCount = 0;
    for (int ii = 0; ii < arrayLength; ii++)
    {
        int iBin = (x[ii]-firstEdge)/binSize;
        if (iBin < 0)
        {
            iBin = 0;
        }
        if (iBin >= numBins)
        {
            iBin = numBins - 1;
        }
        
        if (iBin == idx)
        {
            myBinCount++;
        }
    }
    
    outHist[idx] = myBinCount;
}


void histogram_thread_bin(int arrayLength, const float* x, int numBins, float firstEdge, float lastEdge, int* outHist, int gridSize, int blockSize)
{
    float* d_x;
    int* d_histogram;
    
    checkCudaErrors(hipMalloc(&d_x, arrayLength*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_histogram, numBins*sizeof(int)));
    
    checkCudaErrors(hipMemcpy(d_x, x, arrayLength*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_histogram, 0, numBins*sizeof(int)));
    
    histogram_thread_bin_kernel<<<gridSize, blockSize>>>(arrayLength, d_x, numBins, firstEdge, lastEdge, d_histogram);
    checkCudaErrors(hipMemcpy(outHist, d_histogram, numBins*sizeof(float), hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_histogram));
}
