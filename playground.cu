#include "hip/hip_runtime.h"
#include "utils.h"

void myCudaCaller(int arrayLength, float a, const float* x, const float* y, float* out, int gridSize, int blockSize)
{
    float* d_x;
    float* d_y;
    float* d_out;
    
    checkCudaErrors(hipMalloc(&d_x, arrayLength));
    checkCudaErrors(hipMalloc(&d_y, arrayLength));
    checkCudaErrors(hipMalloc(&d_out, arrayLength));
    checkCudaErrors(hipMemcpy(d_x, x, arrayLength, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y, arrayLength, hipMemcpyHostToDevice));
    myCudaKernel<<<gridSize, blockSize>>>(arrayLength, a, d_x, d_y, d_out);
    checkCudaErrors(hipMemcpy(out, d_out, arrayLength, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_out));
    
    checkCudaErrors(hipDeviceSynchronize()); // only if I need it...
//    checkCudaErrors(hipGetLastError()); // another approach
}

__global__
void myCudaKernel(int arrayLength, float a, const float* x, const float* y, float* out)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (index < arrayLength)
    {
        out[index] = a*x[index] + y[index];
    }
}


