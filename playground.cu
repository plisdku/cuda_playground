#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <stdio.h>

__global__
void saxpy_kernel(int arrayLength, float a, const float* x, const float* y, float* out)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (index < arrayLength)
    {
        out[index] = a*x[index] + y[index];
    }
}

void saxpy(int arrayLength, float a, const float* x, const float* y, float* out, int gridSize, int blockSize)
{
    float* d_x;
    float* d_y;
    float* d_out;
    
    checkCudaErrors(hipMalloc(&d_x, arrayLength));
    checkCudaErrors(hipMalloc(&d_y, arrayLength));
    checkCudaErrors(hipMalloc(&d_out, arrayLength));
    checkCudaErrors(hipMemcpy(d_x, x, arrayLength, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y, arrayLength, hipMemcpyHostToDevice));
    saxpy_kernel<<<gridSize, blockSize>>>(arrayLength, a, d_x, d_y, d_out);
    checkCudaErrors(hipMemcpy(out, d_out, arrayLength, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_out));
    
    checkCudaErrors(hipDeviceSynchronize()); // only if I need it...
//    checkCudaErrors(hipGetLastError()); // another approach
}




// Prefix sum is an inclusive scan: Hillis & Steele.  This is the easy algorithm.
// Exclusive scan starts with a zero: Blelloch scan.  This is the crazy algorithm.

// Hillis & Steele scan
__global__
void prefix_sum_kernel(int arrayLength, float* data)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx >= arrayLength)
    {
        return;
    }
    
    for (int distToLeft = 1; idx - distToLeft >= 0; distToLeft *= 2)
    {
        float tmp = data[idx] + data[idx - distToLeft];
        __syncthreads();
        
        data[idx] = tmp;
        __syncthreads();
        
        // can i possibly really need TWO syncthreads calls per iteration?
    }
}

void prefix_sum(int arrayLength, const float* x, float* out, int gridSize, int blockSize)
{
    float* d_out;
    
    checkCudaErrors(hipMalloc(&d_out, arrayLength*sizeof(float)));
    checkCudaErrors(hipMemcpy(d_out, x, arrayLength*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());
    prefix_sum_kernel<<<gridSize, blockSize>>>(arrayLength, d_out);
    checkCudaErrors(hipMemcpy(out, d_out, arrayLength*sizeof(float), hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
}

